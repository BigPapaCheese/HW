// Name:
// Vector Dot product on many block 
// nvcc HW9.cu -o temp
/*
 What to do:
 This code is the solution to HW8. It finds the dot product of vectors that are smaller than the block size.
 Extend this code so that it uses many blocks and many threads and can find the dot product of any vector length.
 Use shared memory in your blocks to speed up your code.
 You will have to do the final reduction on the CPU.
 Set your thread count to 200. Set N to different values to check your code.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 11231 // Length of the vector
#define B 200
// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void dotProductCPU(float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	int gs = ((N+B-1)/B) ;
	BlockSize.x = 200;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = (gs);
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(3*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	
	for(int id = 1; id < n; id++)
	{ 
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *c, int n)
{
	int k = threadIdx.x;
	__shared__ float s[B];
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if (id<n)
	{
	 s[k] = a[id] * b[id];
	
	__syncthreads();
	int i = n/blockDim.x;
	int j = blockDim.x;
	int m = n%blockDim.x;
	
	if (blockIdx.x<i)
	{
		while(j>1)
		{
		if(k< j/2)
		{
		s[k] += s[j-1-k];
		}
		j=(j+1)/2;
		__syncthreads();
		}
	}
		if(blockIdx.x == i)
		{
			while(m>1)
		{
		if(k< m/2)
		{
			s[k] += s[m-1-k];
		}
		m=(m+1)/2;
		__syncthreads();
	}
	
	}
		if(k==0)
		{
			c[blockIdx.x] = s[0];
		}
	}
}


// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
	double percentError;
	
	percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU);
	free(B_CPU);
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	//float localC_CPU, localC_GPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, (N)*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	int u = N/BlockSize.x;
	for (int j=1;j<=u;j++)
	{
		C_CPU[0] +=C_CPU[j];
	}

	DotGPU = C_CPU[0]; // C_GPU was copied into C_CPU.
	
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}


